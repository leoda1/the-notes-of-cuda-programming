#include <hip/hip_runtime.h>

#define N 32*1024*1024
#define BLOCK_SIZE 256

__global__ void reduce_baseline (float * g_idata, float * g_odata) {
    __shared__ float sdata[BLOCK_SIZE];

    unsigned int tid = threadIdx.x;
    unsigned int idx = blockDim.x * blockIdx.x + tid;
    sdata[tid] = g_idata[idx];
    __syncthreads();

    // do reduction in shared memory
    for (unsigned int s = 1; s < blockDim.x; s *= 2) {
        if (tid % (2 * s) == 0) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }
    if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}

int main() {
    float *input_host = (float*)malloc(N*sizeof(float));
    float *input_device;
    hipMalloc((void **)&input_device, N*sizeof(float));
    for (int i = 0; i < N; i++) input_host[i] = 2.0;
    hipMemcpy(input_device, input_host, N*sizeof(float), hipMemcpyHostToDevice);

    int32_t block_num = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;
    float *output_host = (float*)malloc((N / BLOCK_SIZE) * sizeof(float));
    float *output_device;
    hipMalloc((void **)&output_device, (N / BLOCK_SIZE) * sizeof(float));
    
    dim3 grid(N / BLOCK_SIZE, 1);
    dim3 block(BLOCK_SIZE, 1);
    reduce_baseline<<<grid, block>>>(input_device, output_device);
    hipMemcpy(output_host, output_device, block_num * sizeof(float), hipMemcpyDeviceToHost);
    return 0;
}