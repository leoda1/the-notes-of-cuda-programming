#include "hip/hip_runtime.h"
#include<iostream>
#include "common.cuh"

/*
__global__ void addGPU(float* a, float* b, float* c, const int N)
{
    const int bid = blockIdx.x;
    const int tid = threadIdx.x;
    const int idx = bid * blockDim.x + tid;
    c[idx] = a[idx] + b[idx];
}

void initialData(float* addr, int elemCount)
{
    for (int i = 0; i < elemCount; i++)
    {
        addr[i] = (float)(rand() & 0xff) / 10.f;
    }
    return;
}

int main()
{
    setGPU();

    int iElemCount = 512;
    size_t stByteCount = iElemCount * sizeof(float);

    float *a, *b, *c;
    a = (float*)malloc(stByteCount);
    b = (float*)malloc(stByteCount);
    c = (float*)malloc(stByteCount);
    if(a != NULL && b != NULL && c != NULL)
    {
        memset(a, 0, stByteCount);
        memset(b, 0, stByteCount);
        memset(c, 0, stByteCount);
    }
    else
    {
        std::cout << "Memory allocation failed." << std::endl;
        exit(-1);
    }

    float *d_a, *d_b, *d_c;
    hipMalloc((float**)&d_a, stByteCount);
    hipMalloc((float**)&d_b, stByteCount);
    hipMalloc((float**)&d_c, stByteCount);
    if(d_a != NULL && d_b != NULL && d_c != NULL)
    {
        hipMemset(d_a, 0, stByteCount);
        hipMemset(d_b, 0, stByteCount);
        hipMemset(d_c, 0, stByteCount);
    }
    else
    {
        std::cout << "Memory allocation on GPU failed." << std::endl;
        free(a);
        free(b);
        free(c);
        exit(-1);
    }
    srand(666);
    initialData(a, iElemCount);
    initialData(b, iElemCount);
    
    hipMemcpy(d_a, a, stByteCount, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, stByteCount, hipMemcpyHostToDevice);
    hipMemcpy(d_c, c, stByteCount, hipMemcpyDeviceToHost);

    dim3 block(32);
    dim3 grid(iElemCount / 32);

    addGPU<<<grid, block>>>(d_a, d_b, d_c, iElemCount);
    hipDeviceSynchronize();

    hipMemcpy(c, d_c, stByteCount, hipMemcpyDeviceToHost);

    for (int i = 0; i < iElemCount; i++)
    {
        std::cout << "a[" << i << "] = " << a[i] << " ";
        std::cout<<"b[" << i << "] = " << b[i] << " ";
        std::cout<<"c[" << i << "] = " << c[i] << std::endl;
    }

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    free(a);
    free(b);
    free(c);
    return 0;
}
*/

int main()
{
    //1.分配主机内存，并初始化
    float *fpHost_A;
    fpHost_A = (float *)malloc(4);
    memset(fpHost_A, 0, 4);//初始化为0

    //2.分配设备内存

    float *fpDev_A;
    hipError_t err = ErrorCheck(hipMalloc((float**)&fpDev_A, 4), __FILE__, __LINE__);
    hipMemset(fpDev_A, 0, 4);//*fpDev_A = 0;

    //3.拷贝数据到设备内存
    ErrorCheck(hipMemcpy(fpDev_A, fpHost_A, 4, hipMemcpyHostToDevice), __FILE__, __LINE__);

    //释放主机和设备内存
    free(fpHost_A);
    ErrorCheck(hipFree(fpDev_A), __FILE__, __LINE__);

    ErrorCheck(hipDeviceReset(), __FILE__, __LINE__);


    return 0;  
}