
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 1024
#define THREADS_PER_BLOCK 256

__global__ void vecadd (float *A, float *B, float *C) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx < N) {
        C[idx] = A[idx] + B[idx];
    }
}

__global__ void vecaddCoarsened(float *A, float *B, float *C) {
    int idx = (blockDim.x * blockIdx.x + threadIdx.x) * 2;
    if (idx < N) {
        C[idx] = A[idx] + B[idx];
    }
    if (idx + 1 < N) {
        C[idx + 1] = A[idx + 1] + B [idx + 1];
    }
    
}

void random_init (float *data, int size) {
    for (int i = 0; i < size; i ++ ) {
        data[i] = rand() / (float)RAND_MAX;
    }
}

int main() {
    float *a,   *b,   *c;
    float *d_a, *d_b, *d_c;
    int size = N * sizeof(float);

    hipMalloc((void**)&d_a, size);
    hipMalloc((void**)&d_b, size);
    hipMalloc((void**)&d_c, size);

    a = (float *)malloc(size); random_init(a, N);
    b = (float *)malloc(size); random_init(b, N);
    c = (float *)malloc(size);

    hipEvent_t start, stop, startCoarsened, stopCoarsend;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventCreate(&startCoarsened);
    hipEventCreate(&stopCoarsend);

    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    //warm up
    vecadd<<<(N + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, THREADS_PER_BLOCK >>>(d_a, d_b, d_c);
    hipDeviceSynchronize();
    vecaddCoarsened<<<(N + 2 * THREADS_PER_BLOCK - 1) / (2 * THREADS_PER_BLOCK), THREADS_PER_BLOCK >>>(d_a, d_b, d_c);
    hipDeviceSynchronize();

    //start kernel 1
    hipEventRecord(start);
    vecadd<<<(N + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, THREADS_PER_BLOCK >>>(d_a, d_b, d_c);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    //start kernel 2
    hipEventRecord(startCoarsened);
    vecaddCoarsened<<<(N + 2 * THREADS_PER_BLOCK - 1) / (2 * THREADS_PER_BLOCK), THREADS_PER_BLOCK >>>(d_a, d_b, d_c);
    hipEventRecord(stopCoarsend);
    hipEventSynchronize(stopCoarsend);
    
    float mscends = 0, mscends_coarsened = 0;
    hipEventElapsedTime(&mscends, start, stop);
    hipEventElapsedTime(&mscends_coarsened, startCoarsened, stopCoarsend);
    printf ("vecadd time: %.6f ms\n", mscends);
    printf ("vecadd with coarsened: %.6f ms\n", mscends_coarsened);

    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

    hipFree(d_a); hipFree(d_b); hipFree(d_c);
    free(a); free(b); free(c);

    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipEventDestroy(startCoarsened);
    hipEventDestroy(stopCoarsend);
    return 0;
    
}