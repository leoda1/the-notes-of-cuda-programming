#include "hip/hip_runtime.h"
/******************************************************************
 * Author      : Da Liu
 * Date        : 2024-07-29
 * File Name   : GPU_Cache.cu
 * Description : 测试   L1 Cache能否使用与L2 Cache大小
 *****************************************************************/

#include <iostream>
#include <hip/hip_runtime.h>
#include "common.cuh"

__global__ void kernel() {}


int main() {
    int device_id = 0;
    hipDeviceProp_t device_prop;
    ErrorCheck(hipGetDeviceProperties(&device_prop, device_id), __FILE__, __LINE__);
    printf("Device %d: %s\n", device_id, device_prop.name);

    if (device_prop.globalL1CacheSupported){
        std::cout << "L1 Cache is supported" << std::endl;
    }
    else{
        std::cout << "L1 Cache is not supported" << std::endl;
    }
    std::cout << "L2 Cache size: " << device_prop.l2CacheSize / (1024 * 1024) << " MB" << std::endl;

    dim3 block(1);
    dim3 grid(1);
    kernel<<<grid, block>>>();

    ErrorCheck(hipDeviceSynchronize(), __FILE__, __LINE__);
    ErrorCheck(hipDeviceReset(), __FILE__, __LINE__);

    return 0;
}

/***********************************************************************
PS C:\Users\22681\Desktop\project\cudalearn> ./build/Debug/GPU_Cache.exe      
Device 0: NVIDIA GeForce RTX 4060 Laptop GPU
L1 Cache is supported
L2 Cache size: 32 MB
 **********************************************************************/