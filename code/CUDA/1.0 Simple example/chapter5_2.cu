//kernal defination

#include <hip/hip_runtime.h>
#include <iostream>

#define N 3  //matrix size
__global__ void MatAdd(float A[N][N], float B[N][N], float C[N][N]) {
    int i = threadIdx.x;
    int j = threadIdx.y;
    C[i][j] = A[i][j] + B[i][j];
}

int main() {
    //Kernal invocation with ONE block of N*N*1 threads
    float A[N][N] = { {1, 2, 3}, {4, 5, 6}, {7, 8, 9} };
    float B[N][N] = { {9, 8, 7}, {6, 5, 4}, {3, 2, 1} };
    float C[N][N];
    
    int gridDim = 1;//定义网格中有一个block
    dim3 blockDim(N, N);//定义一个block中有N*N个线程
    MatAdd<<<gridDim, blockDim>>>(A, B, C);
    
    hipDeviceSynchronize();//CUDA同步函数，执行完核函数后继续执行主机程序
    std::cout << "Matrix C:" << std::endl;
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            std::cout << C[i][j] << " ";
        }
        std::cout << std::endl;
    }
    return 0;
}