#include "hip/hip_runtime.h"
/******************************************************************
 * Author      : Da Liu
 * Date        : 2024-07-29
 * File Name   : constant_memory.cu
 * Description : 常量内存的使用
 *****************************************************************/
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "common.cuh"
__constant__ float c_data;
__constant__ float c_data2 = 6.6f;

__global__ void kernel_1(void)
{
    printf("constant data c_data = %.2f.\n", c_data);
}

int main(int argc, char** argv)
{
    int device_id = 0;
    hipDeviceProp_t device_prop;
    ErrorCheck(hipGetDeviceProperties(&device_prop, device_id), __FILE__, __LINE__);
    printf("Device %d: %s\n", device_id, device_prop.name);

    float h_data = 3.3f;
    ErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(c_data), &h_data, sizeof(float)), __FILE__, __LINE__);

    dim3 block(1);
    dim3 grid(1);
    kernel_1<<<grid, block>>>();
    ErrorCheck(hipDeviceSynchronize(), __FILE__, __LINE__);
    ErrorCheck(hipMemcpyFromSymbol(&h_data, HIP_SYMBOL(c_data2), sizeof(float)), __FILE__, __LINE__);
    printf("constant data h_data = %.2f.\n", h_data);

    ErrorCheck(hipDeviceReset(), __FILE__, __LINE__);
    return 0;
}

/******************************************************************
PS C:\Users\22681\Desktop\project\cudalearn> ./build/Debug/constant_memory.exe
Device 0: NVIDIA GeForce RTX 4060 Laptop GPU
constant data c_data = 3.30.
constant data h_data = 6.60.
 *****************************************************************/