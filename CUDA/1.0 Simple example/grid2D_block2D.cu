#include "hip/hip_runtime.h"
/******************************************************************
 * Author: Da Liu
 * Date: 2024-07-25
 * File: grid2D_block2D.cu
 * Description: 组织线程模型：二维网格二维线程块计算二维矩阵加法.
 *****************************************************************/

#include <stdio.h>
#include "common.cuh"

__global__ void add_matrix(int *a, int *b, int *c, const int nx, const int ny) 
{
    /* 2D网格二维/一维线程块计算二维矩阵加法 
    int ix = threadIdx.x + blockIdx.x * blockDim.x;
    int iy = threadIdx.y + blockIdx.y * blockDim.y;
    unsigned int idx = iy * nx + ix;
    if (ix < nx && iy < ny) {
        c[idx] = a[idx] + b[idx];
    } */

   /* 1D网格一维线程块计算二维矩阵加法 */
    int ix= blockIdx.x * blockDim.x + threadIdx.x;;
    if (ix < nx)
    {
        for (int iy = 0; iy < ny; iy ++)
        {
            int idx = iy * nx + ix;
            c[idx] = a[idx] + b[idx];
        }
    }

}

int main()
{
    setGPU();                                           //设置GPU设备
    
    int nx = 16, ny = 8;                                //矩阵大小
    int nxy = nx * ny;                                  //矩阵元素个数
    size_t stBytesCount = nxy * sizeof(int);            //矩阵元素字节数

    int *ipHost_A, *ipHost_B, *ipHost_C;                //主机内存
    ipHost_A = (int* )malloc(stBytesCount);
    ipHost_B = (int* )malloc(stBytesCount);
    ipHost_C = (int* )malloc(stBytesCount);
    if (ipHost_A != NULL && ipHost_B != NULL && ipHost_C != NULL)
    {
        for(int i = 0; i < nxy; i++)
        {
            ipHost_A[i] = i;                             //矩阵A元素初始化为0到nxy-1
            ipHost_B[i] = i + 1;                         //矩阵B元素初始化为A元素+1
        }
        memset(ipHost_C, 0, stBytesCount);               //初始化矩阵C为0
    }
    else
    {
        printf("Memory allocation failed!\n");
        exit(-1);
    }
    int *ipDevice_A, *ipDevice_B, *ipDevice_C;           //设备内存
    ErrorCheck(hipMalloc((int**)&ipDevice_A, stBytesCount), __FILE__, __LINE__);
    ErrorCheck(hipMalloc((int**)&ipDevice_B, stBytesCount), __FILE__, __LINE__);
    ErrorCheck(hipMalloc((int**)&ipDevice_C, stBytesCount), __FILE__, __LINE__);
    if (ipDevice_A != NULL && ipDevice_B != NULL && ipDevice_C != NULL)
    {
        ErrorCheck(hipMemcpy(ipDevice_A, ipHost_A, stBytesCount, hipMemcpyHostToDevice), __FILE__, __LINE__);
        ErrorCheck(hipMemcpy(ipDevice_B, ipHost_B, stBytesCount, hipMemcpyHostToDevice), __FILE__, __LINE__);
        ErrorCheck(hipMemcpy(ipDevice_C, ipHost_C, stBytesCount, hipMemcpyHostToDevice), __FILE__, __LINE__);
    }
    else
    {
        printf("Device Memory copy failed!\n");
        free(ipHost_A);
        free(ipHost_B);
        free(ipHost_C);
        exit(1);
    }

    dim3 block(4, 1);
    // dim3 grid((nx + block.x - 1) / block.x, (ny + block.y - 1) / block.y);       //2Dgrid2Dblock
    // dim3 grid((nx + block.x - 1) / block.x, ny);                                 //1Dgrid1Dblock
    dim3 grid((nx + block.x -1) / block.x, 1);
    printf("Grid: %d, %d\n", grid.x, grid.y);
    printf("Block: %d, %d\n", block.x, block.y);

    add_matrix<<<grid, block>>>(ipDevice_A, ipDevice_B, ipDevice_C, nx, ny);
    ErrorCheck(hipMemcpy(ipHost_C, ipDevice_C, stBytesCount, hipMemcpyDeviceToHost), __FILE__, __LINE__);
    for (int i = 0; i < 10; i++)
    {
        printf("id = %d, matrix_A = %d, matrix_B = %d, matrix_C = %d\n", i + 1, ipHost_A[i], ipHost_B[i], ipHost_C[i]);
    }
    free(ipHost_A);
    free(ipHost_B);
    free(ipHost_C);
    ErrorCheck(hipFree(ipDevice_A), __FILE__, __LINE__);
    ErrorCheck(hipFree(ipDevice_B), __FILE__, __LINE__);
    ErrorCheck(hipFree(ipDevice_C), __FILE__, __LINE__);
    return 0;
}

/******************************************************************
 * Number of CUDA-capable devices found: 1
Device 0 set successfully.
Grid: 4, 1
Block: 4, 1
id = 1, matrix_A = 0, matrix_B = 1, matrix_C = 1
id = 2, matrix_A = 1, matrix_B = 2, matrix_C = 3
id = 3, matrix_A = 2, matrix_B = 3, matrix_C = 5
id = 4, matrix_A = 3, matrix_B = 4, matrix_C = 7
id = 5, matrix_A = 4, matrix_B = 5, matrix_C = 9
id = 6, matrix_A = 5, matrix_B = 6, matrix_C = 11
id = 7, matrix_A = 6, matrix_B = 7, matrix_C = 13
id = 8, matrix_A = 7, matrix_B = 8, matrix_C = 15
id = 9, matrix_A = 8, matrix_B = 9, matrix_C = 17
id = 10, matrix_A = 9, matrix_B = 10, matrix_C = 19
 *****************************************************************/