#include "hip/hip_runtime.h"
/******************************************************************
 * Author: Da Liu
 * Date: 2024-07-27
 * File: global_memory.cu
 * Description: 静态全局变量的使用.
 *****************************************************************/

#include <hip/hip_runtime.h>
#include <stdio.h>
#include "common.cuh"

__device__ int d_x = 1;
__device__ int d_y[2]; 

__global__ void kernel(void)
{
    d_y[0] += d_x;
    d_y[1] += d_x;
    printf("d_x = %d, d_y[0] = %d, d_y[1] = %d\n", d_x, d_y[0], d_y[1]);
}

int main(int argc, char** argv)
{
    int device_id = 0;
    hipDeviceProp_t device_prop;
    ErrorCheck(hipGetDeviceProperties(&device_prop, device_id), __FILE__, __LINE__);

    //std::cout << "运行时GPU设备:" << device_prop.name << std::endl;
    printf("the device of GPU: %s\n", device_prop.name);

    int h_y[2] = {10, 20};
    ErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(d_y), h_y, sizeof(int) * 2));

    dim3 block(1);
    dim3 grid(1);
    kernel<<<grid, block>>>();
    ErrorCheck(hipDeviceSynchronize(), __FILE__, __LINE__);
    ErrorCheck(hipMemcpyFromSymbol(h_y, HIP_SYMBOL(d_y), sizeof(int) * 2), __FILE__, __LINE__);
    printf("h_y[0] = %d, h_y[1] = %d\n", h_y[0], h_y[1]);

    ErrorCheck(hipDeviceReset(), __FILE__, __LINE__);
    return 0;
}