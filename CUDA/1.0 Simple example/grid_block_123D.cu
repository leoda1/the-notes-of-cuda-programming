// #include <cuda_runtime.h>

#include <hip/hip_runtime.h>
#include <stdio.h>


__global__ void print_idx_kernel(){
    printf("block idx: (%3d, %3d, %3d), thread idx: (%3d, %3d, %3d)\n",
         blockIdx.z, blockIdx.y, blockIdx.x,
         threadIdx.z, threadIdx.y, threadIdx.x);
}

__global__ void print_dim_kernel(){
    printf("grid dimension: (%3d, %3d, %3d), block dimension: (%3d, %3d, %3d)\n",
         gridDim.z, gridDim.y, gridDim.x,
         blockDim.z, blockDim.y, blockDim.x);
}

__global__ void print_thread_idx_per_block_kernel(){
    int index = threadIdx.z * blockDim.x * blockDim.y + \
              threadIdx.y * blockDim.x + \
              threadIdx.x;

    printf("block idx: (%3d, %3d, %3d), thread idx: %3d\n",
         blockIdx.z, blockIdx.y, blockIdx.x,
         index);
}

__global__ void print_thread_idx_per_grid_kernel(){
    int bSize  = blockDim.z * blockDim.y * blockDim.x;

    int bIndex = blockIdx.z * gridDim.x * gridDim.y + \
               blockIdx.y * gridDim.x + \
               blockIdx.x;

    int tIndex = threadIdx.z * blockDim.x * blockDim.y + \
               threadIdx.y * blockDim.x + \
               threadIdx.x;

    int index  = bIndex * bSize + tIndex;

    printf("block idx: %3d, thread idx in block: %3d, thread idx: %3d\n", 
         bIndex, tIndex, index);
}

__global__ void print_cord_kernel(){
    int index = threadIdx.z * blockDim.x * blockDim.y + \
              threadIdx.y * blockDim.x + \
              threadIdx.x;

    int x  = blockIdx.x * blockDim.x + threadIdx.x;
    int y  = blockIdx.y * blockDim.y + threadIdx.y;
    int z  = blockIdx.z * blockDim.z + threadIdx.z;

    printf("block idx: (%3d, %3d, %3d), thread idx: %3d, cord: (%3d, %3d, %3d)\n",
         blockIdx.z, blockIdx.y, blockIdx.x,
         index, x, y, z);
}

void print_one_dim(){
    int inputSize = 8;
    int blockDim = 4;
    int gridDim = inputSize / blockDim;

    dim3 block(blockDim);
    dim3 grid(gridDim);
    printf("grid dimension:         %d, block dimension:            %d,\n",
           grid.x,
           block.x);
    // print_idx_kernel<<<grid, block>>>();
    // print_dim_kernel<<<grid, block>>>();
    // print_thread_idx_per_block_kernel<<<grid, block>>>();
    // print_thread_idx_per_grid_kernel<<<grid, block>>>();
    // print_cord_kernel<<<grid, block>>>();

    hipDeviceSynchronize();
}

void print_two_dim(){
    int inputWidth = 4;

    int blockDim = 2;
    int gridDim = inputWidth / blockDim;

    dim3 block(blockDim, blockDim);
    dim3 grid(gridDim, gridDim);
    printf("grid dimension:    (%d, %d), block dimension: (%d, %d)\n",
           grid.y, grid.x,
           block.y, block.x);
    // print_idx_kernel<<<grid, block>>>();
    // print_dim_kernel<<<grid, block>>>();
    // print_thread_idx_per_block_kernel<<<grid, block>>>();
    // print_thread_idx_per_grid_kernel<<<grid, block>>>();
    // print_cord_kernel<<<grid, block>>>();

    hipDeviceSynchronize();
}

void print_three_dim(){
    // 设置网格和块的三维大小
    int depth = 3;    // 深度
    int height = 3;   // 高度
    int width = 3;    // 宽度

    // 块的大小，这里设置为每个维度2个线程
    int blockDim = 2;
    
    // 计算需要的网格大小，确保覆盖所有的数据点
    dim3 block(blockDim, blockDim, blockDim);  // 每个块的维度
    dim3 grid((width + blockDim - 1) / blockDim, 
              (height + blockDim - 1) / blockDim,
              (depth + blockDim - 1) / blockDim);  // 网格的维度
    printf("grid dimension: (%d, %d, %d), block dimension: (%d, %d, %d)\n",
           grid.z, grid.y, grid.x,
           block.z, block.y, block.x);
    // 调用你的kernel函数来打印索引和其他信息
    // 你可以基于需要激活以下kernel中的一个或多个
    // print_idx_kernel<<<grid, block>>>();
    // print_dim_kernel<<<grid, block>>>();
    // print_thread_idx_per_block_kernel<<<grid, block>>>();
    // print_thread_idx_per_grid_kernel<<<grid, block>>>();
    print_cord_kernel<<<grid, block>>>();

    // 确保CUDA操作完成，方便看到打印输出
    hipDeviceSynchronize();
}

int main() {
    /*
    synchronize是同步的意思，有几种synchronize方法：
    cudaDeviceSynchronize: CPU与GPU端完成同步，CPU不执行之后的语句，知道这个语句以前的所有cuda操作结束
    cudaStreamSynchronize: 跟cudaDeviceSynchronize很像，但是这个是针对某一个stream的。只同步指定的stream中的cpu/gpu操作，其他的不管
    cudaThreadSynchronize: 现在已经不被推荐使用的方法
    __syncthreads:         线程块内同步
    */

    // print_one_dim();
    // print_two_dim();
    print_three_dim();

    return 0;
}
